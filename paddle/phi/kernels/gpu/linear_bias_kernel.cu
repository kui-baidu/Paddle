// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/linear_bias_kernel.h"
#include "paddle/phi/backends/dynload/hipblas.h"
#include "paddle/phi/backends/dynload/hipblaslt.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/*
#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11000
#include <hipblaslt.h>
#endif
*/

namespace phi {

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int64_t m,
    int64_t n,
    int64_t k,
    const float* alpha,
    const phi::dtype::float16* A,
    int64_t lda,
    const phi::dtype::float16* B,
    int64_t ldb,
    const float* beta,
    phi::dtype::float16* C,
    int64_t ldc) {
  //return hipblasGemmEx(
  return phi::dynload::hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// BF16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int64_t m,
    int64_t n,
    int64_t k,
    const float* alpha,
    const phi::dtype::bfloat16* A,
    int64_t lda,
    const phi::dtype::bfloat16* B,
    int64_t ldb,
    const float* beta,
    phi::dtype::bfloat16* C,
    int64_t ldc) {
  //return hipblasGemmEx(
  return phi::dynload::hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_16BF,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int64_t m,
    int64_t n,
    int64_t k,
    const float *alpha, /* host pointer */
    const phi::dtype::float16* A,
    int64_t lda,
    const phi::dtype::float16* B,
    int64_t ldb,
    const float *beta, /* host pointer */
    phi::dtype::float16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  //status = hipblasLtMatmul(ltHandle,
  status = phi::dynload::hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int64_t m,
    int64_t n,
    int64_t k,
    const float *alpha, /* host pointer */
    const phi::dtype::bfloat16* A,
    int64_t lda,
    const phi::dtype::bfloat16* B,
    int64_t ldb,
    const float *beta, /* host pointer */
    phi::dtype::bfloat16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = phi::dynload::hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  //status = hipblasLtMatmul(ltHandle,
  status = phi::dynload::hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

#endif


template <typename T, typename Context>
int linear_bias_forward_cuda(const Context& ctx, const DenseTensor& input, const T *weight, const DenseTensor& bias, int64_t in_features, int64_t batch_size, int64_t out_features, DenseTensor* output, void *lt_workspace) {
    hipblasHandle_t handle = ctx.cublas_handle();

    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    const float beta_one       = 1.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    input.data<T>(),
    in_features,
    &beta_zero, /* host pointer */
    output->data<T>(),
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(bias.data<T>()));
#endif
    if (status != 0){
        phi::Copy<Context>(ctx, bias, ctx.GetPlace(), false, output);

        //phi::funcs::CBlas<T>::GEMM(
        status = gemm_bias(
          handle,
          HIPBLAS_OP_T,
          HIPBLAS_OP_N,
          out_features,
          batch_size,
          in_features,
          &alpha,
          weight,
          in_features,
          input.data<T>(),
          in_features,
          &beta_one,
          output->data<T>(),
          out_features);
    }
    return status;
}

template <typename T, typename Context>
void LinearBiasKernel(const Context& ctx, const DenseTensor& input, const DenseTensor& weight, const DenseTensor& bias, DenseTensor* out) {

  std::vector<std::int64_t> input_dims = vectorize(input.dims());
  std::vector<std::int64_t> weight_dims = vectorize(weight.dims());

  int64_t batch_size = input_dims[0];
  int64_t in_features = input_dims[1];
  int64_t out_features = weight_dims[0];

  ctx.template Alloc<T>(out);

  // allocate fixed 4MB workspace for cublaslt for now, and this gets at least 4 MB
  DenseTensor lt_workspace = phi::Empty<T, Context>(ctx, {1 << 22});
  T* lt_workspace_ptr = lt_workspace.data<T>();

    const T* w_ptr = weight.data<T>();
    auto result = linear_bias_forward_cuda<T, Context>(
        ctx,
        input,
        w_ptr,
        bias,
        in_features,
        batch_size,
        out_features,
        out,
        (void*) lt_workspace_ptr);
    PADDLE_ENFORCE_EQ(result, 0, phi::errors::InvalidArgument("linear_bias_forward_cuda failed."));
}

}  // namespace phi

PD_REGISTER_KERNEL(linear_bias,
                   GPU,
                   ALL_LAYOUT,
                   phi::LinearBiasKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
